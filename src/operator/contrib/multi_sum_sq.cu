#include "hip/hip_runtime.h"
/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */

/*!
 *  Copyright (c) 2019 by Contributors
 * \file multi_sum_sq.cu
 * \brief vectorized sums of squares norm over multiple arrays operators
 * \author Clement Fuji Tsang
 */
#include "./multi_sum_sq-inl.h"
#include <hipcub/hipcub.hpp>

#define ILP 4
#define BLOCK_LIMIT 320
#define ARRAY_LIMIT 110

namespace mxnet {
namespace op {

// Shamelessly gotten from:
// https://github.com/NVIDIA/apex/blob/master/csrc/multi_tensor_apply.cuh
// https://github.com/NVIDIA/apex/blob/master/csrc/multi_tensor_l2norm_kernel.cu
// https://github.com/NVIDIA/apex/blob/master/csrc/type_shim.h
template <typename DType>
struct MultiSumSqKernelParam {
  DType* addresses[ARRAY_LIMIT];
  int sizes[ARRAY_LIMIT];
  unsigned char block_to_tensor[BLOCK_LIMIT];
  int block_to_chunk[BLOCK_LIMIT];
};

template<typename DType>
__device__ __forceinline__ DType reduce_block_into_lanes(DType* x,
                                                         DType val,
                                                         int lanes = 1,
                                                         bool share_result = false) {
  int tid = threadIdx.x + threadIdx.y * blockDim.x;
  int blockSize = blockDim.x * blockDim.y;  // blockSize is intended to be a multiple of 32.

  if (blockSize >= 64) {
    x[tid] = val;
    __syncthreads();
  }

  #pragma unroll
  for (int i = (blockSize >> 1); i >= 64; i >>= 1) {
    if (tid < i)
      x[tid] = x[tid] + x[tid+i];
    __syncthreads();
  }

  DType final;

  if (tid < 32) {
    if (blockSize >= 64)
      final = x[tid] + x[tid+32];
    else
      final = val;
    // __SYNCWARP();

    #pragma unroll
    for (int i = 16; i >= lanes; i >>= 1)
      final = final + __shfl_down_sync(0xffffffff, final, i);
  }

  if (share_result) {
    if (tid < lanes)
      x[tid] = final;  // EpilogueOp
    // Make sure the smem result is visible to all warps.
    __syncthreads();
  }

  return final;
}

template<typename DType>
__global__ void MultiSumSqKernel(int chunk_size,
                                 MultiSumSqKernelParam<DType> param,
                                 float* output) {
  int tensor_loc = param.block_to_tensor[blockIdx.x];
  int chunk_idx = param.block_to_chunk[blockIdx.x];
  int n = param.sizes[tensor_loc];

  DType* x = param.addresses[tensor_loc];
  x += chunk_idx * chunk_size;
  n -= chunk_idx * chunk_size;
  __shared__ float vals[512];

  // Non-divergent exit condition for __syncthreads, not necessary here
  float incoming_vals[ILP];
  float val = 0;
  for (int i_start = 0;
       i_start < n && i_start < chunk_size;
       i_start += blockDim.x * ILP) {
    #pragma unroll
    for (int ii = 0; ii < ILP; ++ii) {
      incoming_vals[ii] = 0;
      int i = i_start + threadIdx.x + ii * blockDim.x;
      if (i < n && i < chunk_size)
        incoming_vals[ii] = static_cast<float>(x[i]);
    }

    #pragma unroll
    for (int ii = 0; ii < ILP; ++ii) {
      int i = i_start + threadIdx.x + ii * blockDim.x;
      if (i < n && i < chunk_size)
        val += incoming_vals[ii] * incoming_vals[ii];
    }
  }

  float final = reduce_block_into_lanes(vals, val);
  if (threadIdx.x == 0) {
    atomicAdd(&output[tensor_loc], final);
  }
}

inline void MultiSumSqGPU(const nnvm::NodeAttrs& attrs,
                          const OpContext &ctx,
                          const std::vector<TBlob> &inputs,
                          const std::vector<OpReqType> &req,
                          const std::vector<TBlob> &outputs) {
  const int chunk_size = 32768;
  const int block_size = 512;
  using namespace mxnet_op;
  Stream<gpu>* s = ctx.get_stream<gpu>();
  float* out_ptr = outputs[0].FlatTo2D<gpu, float>(s).dptr_;
  const auto& p = nnvm::get<MultiSumSqParam>(attrs.parsed);
  auto count = p.num_arrays;
  CUDA_CALL(hipMemsetAsync(out_ptr, 0.f, count * sizeof(float),
            mshadow::Stream<gpu>::GetStream(s)));
  MSHADOW_REAL_TYPE_SWITCH(inputs[0].type_flag_, DType, {
    MultiSumSqKernelParam<DType> param;
    int loc_block_info = 0;   // position in param.block_to_tensor and param.block_to_chunck
    int loc_tensor_info = 0;  // position in param.sizes and param.addresses
    int output_offset = 0;    // array index of the first block pointed on by param.addresses
    for (int t = 0; t < count; t++) {  // array index in inputs
      param.sizes[loc_tensor_info] = inputs[t].shape_.Size();
      param.addresses[loc_tensor_info] = inputs[t].FlatTo2D<gpu, DType>(s).dptr_;
      loc_tensor_info++;
      int chunks_this_tensor = (inputs[t].shape_.Size() + chunk_size - 1) / chunk_size;
      for (int chunk = 0; chunk < chunks_this_tensor; ++chunk) {  // array chunk index
        param.block_to_tensor[loc_block_info] = loc_tensor_info - 1;
        param.block_to_chunk[loc_block_info] = chunk;
        loc_block_info++;

        bool tensors_full = (loc_tensor_info == 110 &&
                             chunk == chunks_this_tensor - 1);
        bool blocks_full = (loc_block_info == 320);
        bool last_chunk = (t == count - 1 && chunk == chunks_this_tensor - 1);
        if (tensors_full || blocks_full || last_chunk) {
          MultiSumSqKernel<<<loc_block_info, block_size, 0,
                             mshadow::Stream<gpu>::GetStream(s)>>>(
                             chunk_size, param, out_ptr + output_offset);
          MSHADOW_CUDA_POST_KERNEL_CHECK(MultiSumSqKernel);
          loc_block_info = 0;
          if (chunk == chunks_this_tensor - 1) {  // if you start from a new tensor
            loc_tensor_info = 0;
            output_offset = t + 1;
          } else {  // if you start from the same tensor
            param.sizes[0] = param.sizes[loc_tensor_info-1];
            param.addresses[0] = param.addresses[loc_tensor_info-1];
            loc_tensor_info = 1;
            output_offset = t;
          }
        }
      }
    }
  });
}

NNVM_REGISTER_OP(multi_sum_sq)
.set_attr<FCompute>("FCompute<gpu>", MultiSumSqGPU);

}  // namespace op
}  // namespace mxnet
